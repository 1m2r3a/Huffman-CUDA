#include "hip/hip_runtime.h"
#include <ctime>
#include "hip/hip_runtime.h"
#include ""
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/copy.h>
#include <thrust/sort.h>
#include <thrust/reduce.h>
#include <thrust/iterator/constant_iterator.h>
#include <string>
#include <iostream>
#include <stdio.h>
#include <fstream>
#include <chrono>
#include <queue>
#include <string.h>
#include <vector>
#include <algorithm>
#include <stdlib.h>
#include "HuffmanTreeBuilder.cpp"

using namespace std;
using namespace thrust;

__constant__ HuffmanTable  deviceHuffmanTable[256];

__global__ void mykerneldecompres(myStringer* encode, int encodeSize,HuffmanTable* myStructure,int* myStrSize,myStringer* out,decompresserBinary* binary, int fs){
	int UID= threadIdx.x + blockIdx.x * blockDim.x;

	if(UID < encodeSize  )
	{
		for(int i=0;i<=encode[UID].length ;i++)
		{
			if(i==encode[UID].length){
			switch(encode[UID].position)
			{
			
				case 1:
					binary[UID].ch[binary[UID].length] = (encode[UID].ch[i] & 0x01) ? 1:0;
					binary[UID].length++;
					break;

				case 2:
					binary[UID].ch[binary[UID].length] = (encode[UID].ch[i] & 0x02) ? 1:0;
					binary[UID].length++;
					binary[UID].ch[binary[UID].length] = (encode[UID].ch[i] & 0x01) ? 1:0;
					binary[UID].length++;
					break;

				case 3:
					binary[UID].ch[binary[UID].length] = (encode[UID].ch[i] & 0x04) ? 1:0;
					binary[UID].length++;
					binary[UID].ch[binary[UID].length] = (encode[UID].ch[i] & 0x02) ? 1:0;
					binary[UID].length++;
					binary[UID].ch[binary[UID].length] = (encode[UID].ch[i] & 0x01) ? 1:0;
					binary[UID].length++;
					break;

				case 4:
					binary[UID].ch[binary[UID].length] = (encode[UID].ch[i] & 0x08) ? 1:0;
					binary[UID].length++;
					binary[UID].ch[binary[UID].length] = (encode[UID].ch[i] & 0x04) ? 1:0;
					binary[UID].length++;
					binary[UID].ch[binary[UID].length] = (encode[UID].ch[i] & 0x02) ? 1:0;
					binary[UID].length++;
					binary[UID].ch[binary[UID].length] = (encode[UID].ch[i] & 0x01) ? 1:0;
					binary[UID].length++;
					break;

				case 5:
					binary[UID].ch[binary[UID].length] = (encode[UID].ch[i] & 0x10) ? 1:0;
					binary[UID].length++;
					binary[UID].ch[binary[UID].length] = (encode[UID].ch[i] & 0x08) ? 1:0;
					binary[UID].length++;
					binary[UID].ch[binary[UID].length] = (encode[UID].ch[i] & 0x04) ? 1:0;
					binary[UID].length++;
					binary[UID].ch[binary[UID].length] = (encode[UID].ch[i] & 0x02) ? 1:0;
					binary[UID].length++;
					binary[UID].ch[binary[UID].length] = (encode[UID].ch[i] & 0x01) ? 1:0;
					binary[UID].length++;
					break;

				case 6:
					binary[UID].ch[binary[UID].length] = (encode[UID].ch[i] & 0x20) ? 1:0;
					binary[UID].length++;
					binary[UID].ch[binary[UID].length] = (encode[UID].ch[i] & 0x10) ? 1:0;
					binary[UID].length++;
					binary[UID].ch[binary[UID].length] = (encode[UID].ch[i] & 0x08) ? 1:0;
					binary[UID].length++;
					binary[UID].ch[binary[UID].length] = (encode[UID].ch[i] & 0x04) ? 1:0;
					binary[UID].length++;
					binary[UID].ch[binary[UID].length] = (encode[UID].ch[i] & 0x02) ? 1:0;
					binary[UID].length++;
					binary[UID].ch[binary[UID].length] = (encode[UID].ch[i] & 0x01) ? 1:0;
					binary[UID].length++;
					break;

				case 7:
					binary[UID].ch[binary[UID].length] = (encode[UID].ch[i] & 0x40) ? 1:0;
					binary[UID].length++;
					binary[UID].ch[binary[UID].length] = (encode[UID].ch[i] & 0x20) ? 1:0;
					binary[UID].length++;
					binary[UID].ch[binary[UID].length] = (encode[UID].ch[i] & 0x10) ? 1:0;
					binary[UID].length++;
					binary[UID].ch[binary[UID].length] = (encode[UID].ch[i] & 0x08) ? 1:0; 
					binary[UID].length++;
					binary[UID].ch[binary[UID].length] = (encode[UID].ch[i] & 0x04) ? 1:0; 
					binary[UID].length++;
					binary[UID].ch[binary[UID].length] = (encode[UID].ch[i] & 0x02) ? 1:0;
					binary[UID].length++;
					binary[UID].ch[binary[UID].length] = (encode[UID].ch[i] & 0x01) ? 1:0; 
					binary[UID].length++;
					break;

				case 8:
					binary[UID].ch[binary[UID].length] = (encode[UID].ch[i] & 0x80) ? 1:0; 
					binary[UID].length++;
					binary[UID].ch[binary[UID].length] = (encode[UID].ch[i] & 0x40) ? 1:0;
					binary[UID].length++;
					binary[UID].ch[binary[UID].length] = (encode[UID].ch[i] & 0x20) ? 1:0; 
					binary[UID].length++;
					binary[UID].ch[binary[UID].length] = (encode[UID].ch[i] & 0x10) ? 1:0;
					binary[UID].length++;
					binary[UID].ch[binary[UID].length] = (encode[UID].ch[i] & 0x08) ? 1:0; 
					binary[UID].length++;
					binary[UID].ch[binary[UID].length] = (encode[UID].ch[i] & 0x04) ? 1:0;
					binary[UID].length++;
					binary[UID].ch[binary[UID].length] = (encode[UID].ch[i] & 0x02) ? 1:0;
					binary[UID].length++;
					binary[UID].ch[binary[UID].length] = (encode[UID].ch[i] & 0x01) ? 1:0;
					binary[UID].length++;
					break;

				default:
					printf("Fail",UID,encode[UID].position,encode[UID].length);
					break;
				}
			break;
			}
		
			binary[UID].ch[binary[UID].length] = (encode[UID].ch[i] & 0x80) ? 1:0;
			binary[UID].length++;
			binary[UID].ch[binary[UID].length] = (encode[UID].ch[i] & 0x40) ? 1:0;
			binary[UID].length++;
			binary[UID].ch[binary[UID].length] = (encode[UID].ch[i] & 0x20) ? 1:0;
			binary[UID].length++;
			binary[UID].ch[binary[UID].length] = (encode[UID].ch[i] & 0x10) ? 1:0;
			binary[UID].length++;
			binary[UID].ch[binary[UID].length] = (encode[UID].ch[i] & 0x08) ? 1:0;
			binary[UID].length++;
			binary[UID].ch[binary[UID].length] = (encode[UID].ch[i] & 0x04) ? 1:0;
			binary[UID].length++;
			binary[UID].ch[binary[UID].length] = (encode[UID].ch[i] & 0x02) ? 1:0;
			binary[UID].length++;
			binary[UID].ch[binary[UID].length] = (encode[UID].ch[i] & 0x01) ? 1:0;
			binary[UID].length++;
				
			}

			long p=0, k=0 ,j=0 ,e=0;
			
			for ( p=0; p<binary[UID].length; p++)
			{
				for(k=0;k<myStrSize[0];k++)
				{
					
					for( j=0;j<myStructure[k].len;j++)
					{ 
						if (myStructure[k].code[j] == binary[UID].ch[p+j])
						{
							e++;
						}

						if(e == myStructure[k].len)
						{
							out[UID].ch[out[UID].length]=myStructure[k].c;
							out[UID].length++;  
							p+=j;
							if(p >= binary[UID].length)
								goto jump;
							goto jump;
						}
						if(j == myStructure[k].len-1) 
							e=0;
					}
				}
				jump: e=0;
					

		}
	}
}

__global__ void kernelCompression(char* rawData,int size,int* myStrSize,myStringer* myString, int chunkSize)
{
	
	int tableSize = myStrSize[0];
	int UID= threadIdx.x + blockIdx.x * blockDim.x;

	if(UID < size)
	{
		int o=0;

		for(int j = UID * chunkSize; j<(chunkSize*UID)+chunkSize;j++)
		{
			if(j < size)
				for(int i =0;i<tableSize;i++)
			   {
				if(rawData[j] == deviceHuffmanTable[i].c)
				{
					
					for(int k=0;k< deviceHuffmanTable[i].len;k++)
					{
	
						if(myString[UID].position ==8)
						{
							o++;
							myString[UID].position =0;
							myString[UID].length++;
							myString[UID].ch[o]  = (myString[UID].ch[o]  << 1) | deviceHuffmanTable[i].code[k];
							myString[UID].position++;
						}else{
							myString[UID].ch[o]  = (myString[UID].ch[o]  << 1) | deviceHuffmanTable[i].code[k];
							myString[UID].position++;
						}
					}
					break;			
				}
			 }
		}
	}	
}



int startGPUCompression(const char* inputFilename,const int BLOCK_NUMBER, const int THREAD_NUMBER,const char* outputFilename)
{
	
  clock_t begin = clock();
  int fileSize =0;
  hipError_t cudaReturnValue;
  
  char* device_rawData; //device input containing the raw data

  FILE *f = fopen(inputFilename, "rb");
  if(f == NULL) printf("FILE NOT FOUND");
  string host_rawData;
  
  if(f)
  {
	  
	  fseek(f, 0, SEEK_END);
	  fileSize = ftell(f);
	  printf("-----------------------------------------------------------------------------------------\nfile size is: %d %f\n\n", fileSize,ceil(fileSize/(float)(BLOCK_NUMBER*THREAD_NUMBER)));
	  host_rawData.resize(ftell(f));
	  rewind(f);
	  fread(&host_rawData[0],1,host_rawData.size(),f);
	  fclose(f); 
  }
  
    hipMalloc(&device_rawData, host_rawData.size());
    hipMemcpy(device_rawData, host_rawData.c_str(), host_rawData.size(), hipMemcpyHostToDevice);

    device_vector<char> device_vectorIn(host_rawData.begin(), host_rawData.end());

    thrust::sort(device_vectorIn.begin(), device_vectorIn.end());

    device_vector<char> device_symbol(device_vectorIn.size());
    device_vector<int>  device_symbolFrequency(device_vectorIn.size());

    thrust::pair<device_vector<char>::iterator, device_vector<int>::iterator> temporaryPair = reduce_by_key(device_vectorIn.begin(), device_vectorIn.end(),
	   make_constant_iterator(1), device_symbol.begin(),   device_symbolFrequency.begin());
  
    device_symbol.erase(temporaryPair.first, device_symbol.end());
    device_symbolFrequency.erase(temporaryPair.second, device_symbolFrequency.end());
    hipDeviceSynchronize();
  
	clock_t end_thrust = clock();

	int character_frequency[256];
	char character[256];


	thrust::copy(device_symbolFrequency.begin(),device_symbolFrequency.end(),character_frequency);
	thrust::copy(device_symbol.begin(), device_symbol.end(), character);
    
		
	//Build the tree and the table
	buildHuffmanTree(character,character_frequency,device_symbol.size());
    Node * root = myQueue.top();
    myQueue.pop();

	int binaryCode[256], top = 0;
	buildHuffmanTable(root, binaryCode, top);
	
	cudaReturnValue = hipMalloc((void**) &deviceHuffmanTable, sizeof(HuffmanTable)*myIdx);
	if (cudaReturnValue != hipSuccess)  printf("hipMemcpy failed!");

	cudaReturnValue = hipMemcpyToSymbol(HIP_SYMBOL(deviceHuffmanTable), hostHuffmanTable, sizeof(HuffmanTable)*myIdx );
	if (cudaReturnValue != hipSuccess)  printf("hipMemcpy failed!");
 
	int* d_myStrSize;
	hipMalloc((void**) &d_myStrSize, sizeof(int));
	hipMemcpy(d_myStrSize, &myIdx, sizeof(int),hipMemcpyHostToDevice);

	myStringer* myString;
	hipMalloc((void**)&myString, sizeof(myStringer)*THREAD_NUMBER*BLOCK_NUMBER);
	hipMemset(myString,0,sizeof(myStringer)*THREAD_NUMBER*BLOCK_NUMBER);
	
	cout<<"SIZE: : "<<host_rawData.size()<<" "<<myIdx<<endl;

	int chunkSize=ceil(fileSize/(float)(BLOCK_NUMBER*THREAD_NUMBER));
	
	kernelCompression<<<BLOCK_NUMBER,THREAD_NUMBER>>>(device_rawData,host_rawData.size(),d_myStrSize,myString,chunkSize);

	ofstream myFile (outputFilename, ios::out | ios::binary);

    myFile<<BLOCK_NUMBER<<" "<<THREAD_NUMBER<<endl;

	myFile<<fileSize<<endl<<myIdx<<endl;

	for(int i=0;i<myIdx;i++)
		myFile<<(int)character[i]<<" "<<character_frequency[i]<<" ";

	hipDeviceSynchronize();

	cout<<"End compression\n";
	clock_t end_encode = clock();

	myStringer *host=(myStringer*)malloc(sizeof(myStringer)*BLOCK_NUMBER*THREAD_NUMBER);
	hipMemcpy(host, myString, sizeof(myStringer)*BLOCK_NUMBER*THREAD_NUMBER,hipMemcpyDeviceToHost);
	
	for( int i=0;i<BLOCK_NUMBER*THREAD_NUMBER & i< fileSize ;i++)
		if(host[i].ch!=NULL)
		{
			myFile<<host[i].length<<" "<<host[i].position<<" ";
			 myFile.write (host[i].ch, host[i].length+1);
		}
			
	hipFree(device_rawData);
	hipFree(myString);
	hipFree(d_myStrSize);
	hipFree(device_rawData);
	free(host);

	clock_t end = clock();
    double elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
    cout<<"ALL: " <<elapsed_secs<<endl;
    cout<<"Thrust: "<< double(end_thrust - begin) / CLOCKS_PER_SEC<<endl;
    cout<<"Encode: "<< double(end_encode - begin) / CLOCKS_PER_SEC<<endl;
    cout<<"Chunk size" << chunkSize;
	
	return 0;
}

int startGPUDecompression(const char* inputFileName, const char* outputFileName)
{
	
	clock_t start = clock();
  
	int sizeOfTable =0;
	int fileSize = 0;
	char ch[256];
	int character_frequency[256];
	int BLOCK_NUMBER = 256,THREAD_NUMBER = 32;

	ifstream inputEncode(inputFileName,ios::in | ios::binary);

	if(inputEncode == NULL)
		printf("File not found or in use\n");

	inputEncode>>BLOCK_NUMBER;
	inputEncode>>THREAD_NUMBER;
	inputEncode>>fileSize;
	inputEncode>>sizeOfTable;

	printf("\n------------------------------------\n%d\n",fileSize);

	myStringer *host=(myStringer*)malloc(sizeof(myStringer)*BLOCK_NUMBER*THREAD_NUMBER);

	int temp;
	for(int i = 0 ; i< sizeOfTable;i++)
	{
		inputEncode>>temp;
		inputEncode>>character_frequency[i];
		ch[i]=(char)temp;
	}

	buildHuffmanTree(ch,character_frequency,sizeOfTable);
    Node * root = myQueue.top();
    myQueue.pop();
	int binaryCode[256], top = 0;
	buildHuffmanTable(root, binaryCode, top);
 
	char x[10];

	for(int i = 0 ; i< BLOCK_NUMBER*THREAD_NUMBER && i<fileSize;i++)
	{
		inputEncode>>host[i].length;
		inputEncode>>host[i].position;
		inputEncode.read(x,1);
		inputEncode.read(host[i].ch,host[i].length+1);
	}
	cout<<"Stuff has been read\n";
	clock_t middle = clock();

	
    HuffmanTable *d_Str;
	hipMalloc((void**) &d_Str, sizeof(HuffmanTable)*myIdx);
	hipMemcpy(d_Str, hostHuffmanTable,  sizeof(HuffmanTable)*myIdx,hipMemcpyHostToDevice);

	int* d_myStrSize;
	hipMalloc((void**) &d_myStrSize, sizeof(int));
	hipMemcpy(d_myStrSize, &myIdx, sizeof(int),hipMemcpyHostToDevice);

	myStringer* d_encoded;
	hipMalloc((void**) &d_encoded,sizeof(myStringer)*BLOCK_NUMBER*THREAD_NUMBER);
	hipMemcpy(d_encoded,host,sizeof(myStringer)*BLOCK_NUMBER*THREAD_NUMBER, hipMemcpyHostToDevice);

	myStringer* d_output;
	hipMalloc((void**) &d_output,sizeof(myStringer)*BLOCK_NUMBER*THREAD_NUMBER);
	hipMemset(d_output,0,sizeof(myStringer)*BLOCK_NUMBER*THREAD_NUMBER);

	decompresserBinary* d_bin;
	hipMalloc((void**) &d_bin,sizeof(decompresserBinary)*BLOCK_NUMBER*THREAD_NUMBER);
	hipMemset(d_bin,0,sizeof(decompresserBinary)*BLOCK_NUMBER*THREAD_NUMBER);
	
	mykerneldecompres<<<BLOCK_NUMBER,THREAD_NUMBER>>>(d_encoded,BLOCK_NUMBER*THREAD_NUMBER,d_Str,d_myStrSize,d_output,d_bin,fileSize);

	hipDeviceSynchronize();

	myStringer *host_output=(myStringer*)malloc(sizeof(myStringer)*BLOCK_NUMBER*THREAD_NUMBER);
	memset(host_output,0,(sizeof(myStringer)*BLOCK_NUMBER*THREAD_NUMBER));
	hipMemcpy(host_output, d_output, sizeof(myStringer)*BLOCK_NUMBER*THREAD_NUMBER,hipMemcpyDeviceToHost);
	
	ofstream out(outputFileName,ios::binary);

	for( int i=0;i<BLOCK_NUMBER*THREAD_NUMBER ;i++)
	{
		if(host_output[i].ch!=NULL)
			out<<host_output[i].ch;
	}
	clock_t end = clock();
    
	

	cout<<"Middle : " << double(middle-start)/ CLOCKS_PER_SEC<<endl;
	cout<<"Decompress time: "<<double(end - start) / CLOCKS_PER_SEC;
	
	return 0;
}
